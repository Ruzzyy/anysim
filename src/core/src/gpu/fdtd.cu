#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "core/common/sources.h"
#include "core/common/curl.h"

template <typename float_type>
__global__ void fdtd_update_h_kernel (
    unsigned int nx, unsigned int ny,
    const float_type dx, const float_type dy,
    const float_type * __restrict__ ez,
    const float_type * __restrict__ mh,
    float_type * __restrict__ hx,
    float_type * __restrict__ hy)
{
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int idx = j * nx + i;

  if (j < ny && i < nx)
  {
    const float_type cex = update_curl_ex (i, j, nx, ny, dy, ez);
    const float_type cey = update_curl_ey (i, j, nx, dx, ez);

    // update_h
    hx[idx] -= mh[idx] * cex;
    hy[idx] -= mh[idx] * cey;
  }
}

template <typename float_type>
__global__ void fdtd_update_e_kernel (
    float_type t,
    unsigned int nx,
    unsigned int ny,
    const float_type C0_p_dt,
    const float_type dx,
    const float_type dy,
    const float_type * __restrict__ er,
    const float_type * __restrict__ hx,
    const float_type * __restrict__ hy,
    float_type * __restrict__ dz,
    float_type * __restrict__ ez,

    unsigned int sources_count,
    const float_type * __restrict__ sources_frequencies,
    const unsigned int * __restrict__ sources_offsets)
{
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int idx = j * nx + i;

  if (j < ny && i < nx)
  {
    const float_type chz = update_curl_h (i, j, nx, ny, dx, dy, hx, hy);

    dz[idx] += C0_p_dt * chz; // update d = C0 * dt * curl Hz

    // TODO Extract into separate kernel
    for (unsigned int source_id = 0; source_id < sources_count; source_id++)
      if (sources_offsets[source_id] == idx)
        dz[idx] += calculate_source (t, sources_frequencies[source_id]);

    ez[idx] = dz[idx] / er[idx]; // update e
  }
}

template <typename float_type>
void fdtd_step_gpu (
    float_type t,
    const float_type dt,
    unsigned int nx, unsigned int ny,
    const float_type dx, const float_type dy,
    const float_type *mh,
    const float_type *er,
    float_type *ez,
    float_type *dz,
    float_type *hx,
    float_type *hy,

    unsigned int sources_count,
    const float_type * __restrict__ sources_frequencies,
    const unsigned int * __restrict__ sources_offsets)
{
  constexpr auto C0 = static_cast<float_type> (299792458); /// Speed of light [metres per second]

  dim3 block_size = dim3 (32, 32);
  dim3 grid_size;

  grid_size.x = (nx + block_size.x - 1) / block_size.x;
  grid_size.y = (ny + block_size.y - 1) / block_size.y;

  // TODO Calculate block sizes
  fdtd_update_h_kernel<<<grid_size, block_size>>> (nx, ny, dx, dy, ez, mh, hx, hy);

  // TODO Update source
  fdtd_update_e_kernel<<<grid_size, block_size>>> (t, nx, ny, C0 * dt, dx, dy, er, hx, hy, dz, ez, sources_count, sources_frequencies, sources_offsets);
}

#define GEN_FDTD_INSTANCE_FOR(type)                                                               \
  template void fdtd_step_gpu<type>(type t, const type dt, unsigned int nx, unsigned int ny,      \
                                    const type dx, const type dy, const type *mh, const type *er, \
                                    type *ez, type *dz, type *hx, type *hy,                       \
                                    unsigned int sources_count, const type *sources_frequencies,  \
                                    const unsigned int *sources_offsets);

GEN_FDTD_INSTANCE_FOR (float)
GEN_FDTD_INSTANCE_FOR (double)

#undef GEN_FDTD_INSTANCE_FOR
