#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#include "core/common/sources.h"

template <typename float_type>
__global__ void fdtd_update_h_kernel (
    unsigned int nx, unsigned int ny,
    const float_type dx, const float_type dy,
    const float_type * __restrict__ ez,
    const float_type * __restrict__ mh,
    float_type * __restrict__ hx,
    float_type * __restrict__ hy)
{
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int idx = j * nx + i;

  __shared__ float_type cache[33 * 33];

  const unsigned int si = threadIdx.x;
  const unsigned int sj = threadIdx.y;
  const unsigned int sidx = sj * (blockDim.x + 1) + si;

  if (j < ny && i < nx)
  {
    cache[sidx] = ez[idx];

    if (sj == blockDim.y - 1 || j == ny - 1)
      cache[(sj + 1) * (blockDim.x + 1) + si] = j < ny - 1 ? ez[(j + 1) * nx + i] : ez[0 * nx + i];

    if (si == blockDim.x - 1 || i == nx - 1)
      cache[sidx + 1] = i < nx - 1 ? ez[idx + 1] : ez[j * nx + 0];
  }

  __syncthreads ();

  if (j < ny && i < nx)
  {
    const float_type cez = cache[sidx];
    const float_type cex =  (cache[(sj + 1) * (blockDim.x + 1) + si] - cez) / dy;
    const float_type cey = -(cache[sidx + 1] - cez) / dx;

    // update_h
    hx[idx] -= mh[idx] * cex;
    hy[idx] -= mh[idx] * cey;
  }
}

template <typename float_type>
__device__ static float_type update_curl_h (
    unsigned int i,
    unsigned int j,
    const unsigned int nx,
    const unsigned int ny,
    const float_type dx,
    const float_type dy,
    const float_type * __restrict__ hx,
    const float_type * __restrict__ hy)
{
  // TODO For now assume that only periodic boundary conditions exist
  const unsigned int curr_idx   = (j + 0) * nx + i;
  const unsigned int prev_idx_i = i > 0 ? (j + 0) * nx + i - 1 : j * nx + nx - 1;
  const unsigned int prev_idx_j = j > 0 ? (j - 1) * nx + i     : (ny - 1) * nx + i;

  return (hy[curr_idx] - hy[prev_idx_i]) / dx
       - (hx[curr_idx] - hx[prev_idx_j]) / dy;
}

template <typename float_type>
__global__ void fdtd_update_e_kernel (
    float_type t,
    unsigned int nx,
    unsigned int ny,
    const float_type C0_p_dt,
    const float_type dx,
    const float_type dy,
    const float_type * __restrict__ er,
    const float_type * __restrict__ hx,
    const float_type * __restrict__ hy,
    float_type * __restrict__ dz,
    float_type * __restrict__ ez,

    unsigned int sources_count,
    const float_type * __restrict__ sources_frequencies,
    const unsigned int * __restrict__ sources_offsets)
{
  const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int idx = j * nx + i;

  if (j < ny && i < nx)
  {
    const float_type chz = update_curl_h (i, j, nx, ny, dx, dy, hx, hy);

    dz[idx] += C0_p_dt * chz; // update d = C0 * dt * curl Hz

    // TODO Extract into separate kernel
    for (unsigned int source_id = 0; source_id < sources_count; source_id++)
      if (sources_offsets[source_id] == idx)
        dz[idx] += calculate_source (t, sources_frequencies[source_id]);

    ez[idx] = dz[idx] / er[idx]; // update e
  }
}

template <typename float_type>
void fdtd_step_gpu (
    float_type t,
    const float_type dt,
    unsigned int nx, unsigned int ny,
    const float_type dx, const float_type dy,
    const float_type *mh,
    const float_type *er,
    float_type *ez,
    float_type *dz,
    float_type *hx,
    float_type *hy,

    unsigned int sources_count,
    const float_type * __restrict__ sources_frequencies,
    const unsigned int * __restrict__ sources_offsets)
{
  constexpr auto C0 = static_cast<float_type> (299792458); /// Speed of light [metres per second]

  dim3 block_size = dim3 (32, 32);
  dim3 grid_size;

  grid_size.x = (nx + block_size.x - 1) / block_size.x;
  grid_size.y = (ny + block_size.y - 1) / block_size.y;

  // TODO Calculate block sizes
  fdtd_update_h_kernel<<<grid_size, block_size>>> (nx, ny, dx, dy, ez, mh, hx, hy);

  // TODO Update source
  fdtd_update_e_kernel<<<grid_size, block_size>>> (t, nx, ny, C0 * dt, dx, dy, er, hx, hy, dz, ez, sources_count, sources_frequencies, sources_offsets);
}

#define GEN_FDTD_INSTANCE_FOR(type)                                                               \
  template void fdtd_step_gpu<type>(type t, const type dt, unsigned int nx, unsigned int ny,      \
                                    const type dx, const type dy, const type *mh, const type *er, \
                                    type *ez, type *dz, type *hx, type *hy,                       \
                                    unsigned int sources_count, const type *sources_frequencies,  \
                                    const unsigned int *sources_offsets);

GEN_FDTD_INSTANCE_FOR (float)
GEN_FDTD_INSTANCE_FOR (double)

#undef GEN_FDTD_INSTANCE_FOR
